#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime_api.h>

#include "sha5.cuh"
#include "bytes.cuh"
#include "salsa20.cuh"
#include "C25519.cuh"

__device__  __host__ void dump_hex(const void* data, size_t size) {
  char ascii[17];
  size_t i, j;
  ascii[16] = '\0';
  for (i = 0; i < size; i++) {
      printf("%02X ", ((unsigned char*)data)[i]);
      if (((unsigned char*)data)[i] >= ' ' && ((unsigned char*)data)[i] <= '~') {
          ascii[i % 16] = ((unsigned char*)data)[i];
      } else {
          ascii[i % 16] = '.';
      }
      if ((i+1) % 8 == 0 || i+1 == size) {
          printf(" ");
          if ((i+1) % 16 == 0) {
              printf("|  %s \n", ascii);
          } else if (i+1 == size) {
              ascii[(i+1) % 16] = '\0';
              if ((i+1) % 16 <= 8) {
                  printf(" ");
              }
              for (j = (i+1) % 16; j < 16; ++j) {
                  printf("   ");
              }
              printf("|  %s \n", ascii);
          }
      }
  }
}

#define ZT_IDENTITY_GEN_MEMORY 2097152
#define ZT_ADDRESS_LENGTH 5
#define ZT_IDENTITY_GEN_HASHCASH_FIRST_BYTE_LESS_THAN 17

__device__ static inline void _computeMemoryHardHash(const void *publicKey,unsigned int publicKeyBytes,void *digest,void *genmem)
{
	// Digest publicKey[] to obtain initial digest
  SHA512(digest,publicKey,publicKeyBytes);

	// Initialize genmem[] using Salsa20 in a CBC-like configuration since
	// ordinary Salsa20 is randomly seek-able. This is good for a cipher
	// but is not what we want for sequential memory-hardness.
	//memset(genmem,0,ZT_IDENTITY_GEN_MEMORY);
  memset(genmem,0x00,64); // should really only need to initialize the first 64 bytes...
 	Salsa20 s20(digest,(char *)digest + 32);
  s20.crypt20((char *)genmem,(char *)genmem,64);

  for(unsigned long i=64;i<ZT_IDENTITY_GEN_MEMORY;i+=64) {
		unsigned long k = i - 64;
  //  memcpy((char*)genmem+i,(char *)genmem+k,64);
		*((uint64_t *)((char *)genmem + i)) = *((uint64_t *)((char *)genmem + k));
		*((uint64_t *)((char *)genmem + i + 8)) = *((uint64_t *)((char *)genmem + k + 8));
		*((uint64_t *)((char *)genmem + i + 16)) = *((uint64_t *)((char *)genmem + k + 16));
		*((uint64_t *)((char *)genmem + i + 24)) = *((uint64_t *)((char *)genmem + k + 24));
		*((uint64_t *)((char *)genmem + i + 32)) = *((uint64_t *)((char *)genmem + k + 32));
		*((uint64_t *)((char *)genmem + i + 40)) = *((uint64_t *)((char *)genmem + k + 40));
		*((uint64_t *)((char *)genmem + i + 48)) = *((uint64_t *)((char *)genmem + k + 48));
		*((uint64_t *)((char *)genmem + i + 56)) = *((uint64_t *)((char *)genmem + k + 56)); 
		s20.crypt20((char *)genmem + i,(char *)genmem + i,64);
	}

	// Render final digest using genmem as a lookup table
	for(unsigned long i=0;i<(ZT_IDENTITY_GEN_MEMORY / sizeof(uint64_t));) {
		unsigned long idx1 = (unsigned long) (swapBytes(((uint64_t *)genmem)[i++]) % (64 / sizeof(uint64_t))) ;
		unsigned long idx2 = (unsigned long)(swapBytes(((uint64_t *)genmem)[i++]) % (ZT_IDENTITY_GEN_MEMORY / sizeof(uint64_t)));
		uint64_t tmp = ((uint64_t *)genmem)[idx2];
		((uint64_t *)genmem)[idx2] = ((uint64_t *)digest)[idx1];
		((uint64_t *)digest)[idx1] = tmp;
    s20.crypt20(digest,digest,64);
	}
}

__device__ int memcmp(const void * s1, const void * s2, size_t n) {
  if (n != 0) {
    const unsigned char * p1 = (const unsigned char *)s1, *p2 = (const unsigned char *)s2;
    do {
      if (*p1++ != *p2++)
        return (*--p1 - *--p2);
    } while (--n != 0);
  }
  
  return (0);
}

__device__ bool test(unsigned char * digest, void * targets, size_t len){
  size_t i = 0;
  for(i = 0; i < len; i=i+5){ // overflow, needs check to make sure len is divisible by 5
    if(memcmp((uint8_t *)targets+i, digest+59, 5) == 0 )
      return true;
  }
  return false;
}

__global__
void crack(void * targets, size_t target_len, bool benchmark)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  int block = blockIdx.x;
  int id = index+(block*stride);

  uint64_t max = 0xffffffffff / (gridDim.x*blockDim.x); // key space divided by each of us
  uint64_t mykey = (0xffffffffffffffff / (gridDim.x*blockDim.x)) * id; // this threads starting point
  unsigned char out[64];
  memset(out, 0x00, sizeof(out));

  char * genmem = (char *)malloc(ZT_IDENTITY_GEN_MEMORY);
  C25519::Pair kp;
  uint8_t * priv = (uint8_t *)kp.priv.data;
  memset(priv, 0x00,ZT_C25519_PRIVATE_KEY_LEN);

  uint64_t i = 0;
  uint64_t attempt = i;
  uint64_t key = 0;

  if(benchmark){
    // Run through 5 loops and bail
    for(i = 0; i < 5; i++){
      do {
        key = mykey+attempt;
        // Bump the bruteforcer forward one byte to deal with C25519 clamping 
        memcpy(priv+2, &key, sizeof(uint64_t));
        C25519::calcPubKeys(&kp);
        _computeMemoryHardHash(kp.pub.data, ZT_C25519_PUBLIC_KEY_LEN, out, genmem);
        attempt++;
      } while(!(out[0] < ZT_IDENTITY_GEN_HASHCASH_FIRST_BYTE_LESS_THAN));
    }
  }
  else {
    for(i = 0; i < max; i++){
      do {
        key = mykey+attempt;
        // Bump the bruteforcer forward one byte to deal with C25519 clamping
        memcpy(priv+2, &key, sizeof(uint64_t));
        C25519::calcPubKeys(&kp);
        _computeMemoryHardHash(kp.pub.data, ZT_C25519_PUBLIC_KEY_LEN, out, genmem);
        attempt++;
      } while(!(out[0] < ZT_IDENTITY_GEN_HASHCASH_FIRST_BYTE_LESS_THAN));

      if(test(out, targets, target_len)){
        printf("key: %llu, address: %02hx%02hx%02hx%02hx%02hx\n", key, (uint8_t)out[59], (uint8_t)out[60], (uint8_t)out[61], (uint8_t)out[62], (uint8_t)out[63]);
      }
    }
  }
  free(genmem);
}

int main(int argc, char ** argv)
{

  if(argc<=3) {
    printf("needs thread, block and heap size (in mb) args. ./ztcrack <blocks> <threads> <heap>");
    exit(1);
  }  
 
  int blocks = atoi(argv[1]);  
  int threads = atoi(argv[2]);
  int heap = atoi(argv[3]);

  const size_t malloc_limit = size_t(heap) * size_t(1024) * size_t(1024);
  printf("Setting heap size to %zu\n",malloc_limit);
  hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit); 

  FILE * fp;
  if((fp = fopen("targets.dat", "rb"))== NULL){
      printf("[!] Error: Could not open file targets.dat: %s\n", strerror(errno));
      printf("[!] Running in benchmark mode\n");      
      printf("Running %d threads and %d blocks. Total %d\n", threads, blocks, threads*blocks);
      
      crack<<<blocks, threads>>>(0x00, 0x00, true);
  }
  else {
  
    size_t bufsize;
    void * targets;
    
    if (fseek(fp, 0L, SEEK_END) == 0) {
        long ft = ftell(fp);
        if (ft == -1){
          printf("[!] Error with ftell: %s", strerror(errno));
          return 1;
        }
        else if(ft == 0){ // handle empty file
          printf("empty file");
          return 1;
        }
        bufsize = ft;
        printf("bufsize: %zu\n", bufsize);
        // Go back to the start of the file.
        if (fseek(fp, 0L, SEEK_SET) != 0){
          printf("[!] Error: could not fseek: %s\n", strerror(errno));
          return 1;
        }

        // Read the entire file into memory.
        void * file = malloc(bufsize);
        size_t r = fread(file, 1, bufsize, fp);
        printf("read: %zu\n", r);
      
        if ( ferror( fp ) != 0 ){
          printf("[!] Error: fread: %s\n", strerror(errno));
          return 1;
        }

        hipMallocManaged(&targets, bufsize*sizeof(uint8_t));
        memcpy(targets, file, bufsize);
        
        printf("Running %d threads and %d blocks. Total %d\n", threads, blocks, threads*blocks);

        crack<<<blocks, threads>>>((uint8_t *)targets, bufsize, false);
    }
    else{
      printf("fseek");
      return 1;
    }

    fclose(fp);
  }
  
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; 
  hipProfilerStop();
  return 0;
}