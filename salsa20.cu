#include "hip/hip_runtime.h"
/*
 * Based on public domain code available at: http://cr.yp.to/snuffle.html
 *
 * Modifications and C-native SSE macro based SSE implementation by
 * Adam Ierymenko <adam.ierymenko@zerotier.com>.
 *
 * Since the original was public domain, this is too.
 */

 #include "Salsa20.cuh"
 
 #define ROTATE(v,c) (((v) << (c)) | ((v) >> (32 - (c))))
 #define XOR(v,w) ((v) ^ (w))
 #define PLUS(v,w) ((uint32_t)((v) + (w)))

 #ifdef ZT_NO_TYPE_PUNNING
 // Slower version that does not use type punning
 #define U8TO32_LITTLE(p) ( ((uint32_t)(p)[0]) | ((uint32_t)(p)[1] << 8) | ((uint32_t)(p)[2] << 16) | ((uint32_t)(p)[3] << 24) )
 __device__ static inline void U32TO8_LITTLE(uint8_t *const c,const uint32_t v) { c[0] = (uint8_t)v; c[1] = (uint8_t)(v >> 8); c[2] = (uint8_t)(v >> 16); c[3] = (uint8_t)(v >> 24); }
 #else?*
 // Fast version that just does 32-bit load/store
 #define U8TO32_LITTLE(p) (*((const uint32_t *)((const void *)(p))))
 #define U32TO8_LITTLE(c,v) *((uint32_t *)((void *)(c))) = (v)
 #endif // ZT_NO_TYPE_PUNNING

 __device__ void Salsa20::init(const void *key,const void *iv)
 {
     const char *const constants = "expand 32-byte k";
     const uint8_t *const k = (const uint8_t *)key;
     _state.i[0] = U8TO32_LITTLE(constants + 0);
     _state.i[1] = U8TO32_LITTLE(k + 0);
     _state.i[2] = U8TO32_LITTLE(k + 4);
     _state.i[3] = U8TO32_LITTLE(k + 8);
     _state.i[4] = U8TO32_LITTLE(k + 12);
     _state.i[5] = U8TO32_LITTLE(constants + 4);
     _state.i[6] = U8TO32_LITTLE(((const uint8_t *)iv) + 0);
     _state.i[7] = U8TO32_LITTLE(((const uint8_t *)iv) + 4);
     _state.i[8] = 0;
     _state.i[9] = 0;
     _state.i[10] = U8TO32_LITTLE(constants + 8);
     _state.i[11] = U8TO32_LITTLE(k + 16);
     _state.i[12] = U8TO32_LITTLE(k + 20);
     _state.i[13] = U8TO32_LITTLE(k + 24);
     _state.i[14] = U8TO32_LITTLE(k + 28);
     _state.i[15] = U8TO32_LITTLE(constants + 12);
 }
 
 __device__ void Salsa20::crypt20(const void *in,void *out,unsigned int bytes)
 {
     uint8_t tmp[64];
     const uint8_t *m = (const uint8_t *)in;
     uint8_t *c = (uint8_t *)out;
     uint8_t *ctarget = c;
     unsigned int i;
 
     uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
     uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
 
     if (!bytes)
         return;

     j0 = _state.i[0];
     j1 = _state.i[1];
     j2 = _state.i[2];
     j3 = _state.i[3];
     j4 = _state.i[4];
     j5 = _state.i[5];
     j6 = _state.i[6];
     j7 = _state.i[7];
     j8 = _state.i[8];
     j9 = _state.i[9];
     j10 = _state.i[10];
     j11 = _state.i[11];
     j12 = _state.i[12];
     j13 = _state.i[13];
     j14 = _state.i[14];
     j15 = _state.i[15];
 
     for (;;) {
         if (bytes < 64) {
             for (i = 0;i < bytes;++i)
                 tmp[i] = m[i];
             m = tmp;
             ctarget = c;
             c = tmp;
         }
 
         x0 = j0;
         x1 = j1;
         x2 = j2;
         x3 = j3;
         x4 = j4;
         x5 = j5;
         x6 = j6;
         x7 = j7;
         x8 = j8;
         x9 = j9;
         x10 = j10;
         x11 = j11;
         x12 = j12;
         x13 = j13;
         x14 = j14;
         x15 = j15;
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         // 2X round -------------------------------------------------------------
          x4 = XOR( x4,ROTATE(PLUS( x0,x12), 7));
          x8 = XOR( x8,ROTATE(PLUS( x4, x0), 9));
         x12 = XOR(x12,ROTATE(PLUS( x8, x4),13));
          x0 = XOR( x0,ROTATE(PLUS(x12, x8),18));
          x9 = XOR( x9,ROTATE(PLUS( x5, x1), 7));
         x13 = XOR(x13,ROTATE(PLUS( x9, x5), 9));
          x1 = XOR( x1,ROTATE(PLUS(x13, x9),13));
          x5 = XOR( x5,ROTATE(PLUS( x1,x13),18));
         x14 = XOR(x14,ROTATE(PLUS(x10, x6), 7));
          x2 = XOR( x2,ROTATE(PLUS(x14,x10), 9));
          x6 = XOR( x6,ROTATE(PLUS( x2,x14),13));
         x10 = XOR(x10,ROTATE(PLUS( x6, x2),18));
          x3 = XOR( x3,ROTATE(PLUS(x15,x11), 7));
          x7 = XOR( x7,ROTATE(PLUS( x3,x15), 9));
         x11 = XOR(x11,ROTATE(PLUS( x7, x3),13));
         x15 = XOR(x15,ROTATE(PLUS(x11, x7),18));
          x1 = XOR( x1,ROTATE(PLUS( x0, x3), 7));
          x2 = XOR( x2,ROTATE(PLUS( x1, x0), 9));
          x3 = XOR( x3,ROTATE(PLUS( x2, x1),13));
          x0 = XOR( x0,ROTATE(PLUS( x3, x2),18));
          x6 = XOR( x6,ROTATE(PLUS( x5, x4), 7));
          x7 = XOR( x7,ROTATE(PLUS( x6, x5), 9));
          x4 = XOR( x4,ROTATE(PLUS( x7, x6),13));
          x5 = XOR( x5,ROTATE(PLUS( x4, x7),18));
         x11 = XOR(x11,ROTATE(PLUS(x10, x9), 7));
          x8 = XOR( x8,ROTATE(PLUS(x11,x10), 9));
          x9 = XOR( x9,ROTATE(PLUS( x8,x11),13));
         x10 = XOR(x10,ROTATE(PLUS( x9, x8),18));
         x12 = XOR(x12,ROTATE(PLUS(x15,x14), 7));
         x13 = XOR(x13,ROTATE(PLUS(x12,x15), 9));
         x14 = XOR(x14,ROTATE(PLUS(x13,x12),13));
         x15 = XOR(x15,ROTATE(PLUS(x14,x13),18));
 
         x0 = PLUS(x0,j0);
         x1 = PLUS(x1,j1);
         x2 = PLUS(x2,j2);
         x3 = PLUS(x3,j3);
         x4 = PLUS(x4,j4);
         x5 = PLUS(x5,j5);
         x6 = PLUS(x6,j6);
         x7 = PLUS(x7,j7);
         x8 = PLUS(x8,j8);
         x9 = PLUS(x9,j9);
         x10 = PLUS(x10,j10);
         x11 = PLUS(x11,j11);
         x12 = PLUS(x12,j12);
         x13 = PLUS(x13,j13);
         x14 = PLUS(x14,j14);
         x15 = PLUS(x15,j15);
 
         U32TO8_LITTLE(c + 0,XOR(x0,U8TO32_LITTLE(m + 0)));
         U32TO8_LITTLE(c + 4,XOR(x1,U8TO32_LITTLE(m + 4)));
         U32TO8_LITTLE(c + 8,XOR(x2,U8TO32_LITTLE(m + 8)));
         U32TO8_LITTLE(c + 12,XOR(x3,U8TO32_LITTLE(m + 12)));
         U32TO8_LITTLE(c + 16,XOR(x4,U8TO32_LITTLE(m + 16)));
         U32TO8_LITTLE(c + 20,XOR(x5,U8TO32_LITTLE(m + 20)));
         U32TO8_LITTLE(c + 24,XOR(x6,U8TO32_LITTLE(m + 24)));
         U32TO8_LITTLE(c + 28,XOR(x7,U8TO32_LITTLE(m + 28)));
         U32TO8_LITTLE(c + 32,XOR(x8,U8TO32_LITTLE(m + 32)));
         U32TO8_LITTLE(c + 36,XOR(x9,U8TO32_LITTLE(m + 36)));
         U32TO8_LITTLE(c + 40,XOR(x10,U8TO32_LITTLE(m + 40)));
         U32TO8_LITTLE(c + 44,XOR(x11,U8TO32_LITTLE(m + 44)));
         U32TO8_LITTLE(c + 48,XOR(x12,U8TO32_LITTLE(m + 48)));
         U32TO8_LITTLE(c + 52,XOR(x13,U8TO32_LITTLE(m + 52)));
         U32TO8_LITTLE(c + 56,XOR(x14,U8TO32_LITTLE(m + 56)));
         U32TO8_LITTLE(c + 60,XOR(x15,U8TO32_LITTLE(m + 60)));
 
         if (!(++j8)) {
             ++j9;
             /* stopping at 2^70 bytes per nonce is user's responsibility */
         }
 
         if (bytes <= 64) {
             if (bytes < 64) {
                 for (i = 0;i < bytes;++i)
                     ctarget[i] = c[i];
             }

             _state.i[8] = j8;
             _state.i[9] = j9;
 
             return;
         }
 
         bytes -= 64;
         c += 64;
         m += 64;
     }
 }